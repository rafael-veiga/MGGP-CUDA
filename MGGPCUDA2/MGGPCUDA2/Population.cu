#include <iostream>
#include "Population.h"
#include "Subject.h"
#include "Gramatica.h"

Population::Population(Population* p) {
	for (int i = 0; i < h_conf->popSize; i++) {
		//        pop[i] = new Subject(objectives);
		//        p->pop[i]->tree->print();
		pop.push_back(new Subject());
		pop[i]->tree->setTree(p->pop[i]->tree->root->clone(NULL));
	}
};

Population::Population() {
	for (int i = 0; i < h_conf->popSize; i++) {
		//        pop[i] = new Subject(gram->criaArvExp(), objectives);
		pop.push_back(new Subject(h_gram->criaArvExp()));
	}
};

Population::~Population() {
	pop.clear();
	//    for(int i = 0; i < conf->popSize; i++){
	////        delete pop[i];
	//        pop.pop_back();
	//    }
};

void Population::print() {
	//    double v[] = {1.0};
	for (int i = 0; i < h_conf->popSize; i++) {
		//        pop[i]->tree->print();
		cout << pop[i]->fitness << " ";
	}
	cout << endl;
};
