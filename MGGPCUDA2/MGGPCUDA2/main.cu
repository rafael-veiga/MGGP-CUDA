#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <iostream>
#include <ctime>
#include <sstream>
#include <iomanip>
//#include <string>

//#include "Datamaker.h"
#include "No.h"
#include "Tree.h"
#include "Database.h"
#include "Gramatica.h"
#include "Configures.h"
#include "Operators.h"
#include "Search.h"

using namespace std;
Configures* h_conf;
Configures* d_conf;
/*
*
*/
__global__ void teste(Database* d_dados) {
	d_dados->countVar = 20;
}

int main(int argc, char** args) {

	// imprimindo argumentos
	//    for(int i=0; i<argc;i++){
	//    cout << " " << args[i];    
	//    }
	//    cout << endl;
	//string nome_saida = args[8];    
	freopen(args[8], "w", stdout);

	int seed = atoi(args[1]);
	string gramatica = args[2];
	string dados = args[3];
	string grupo = args[4];
	int geracoes = atoi(args[5]);
	int populacao = atoi(args[6]);
	int altura = atoi(args[7]);
	//int complexidade = atoi(args[8]); //0=high 1=terminals

	srand(seed);
	cout.precision(7);

	//configuracoes
	h_conf = new Configures();
	h_conf->treeHigh = altura;
	h_conf->popSize = populacao;
	h_conf->iterations = geracoes;
	h_conf->leastSquare = 1;
	h_conf->elitism = 0.1;
	h_conf->mono = 1; // 0 = monobjetivo;  1 = multiobjetivo
					  //conf->complexity = complexidade; // 0 = high 1 = terminals
	h_conf->complexity = 1;
	//gramatica
	gram = new Gramatica(gramatica);
	// cout<< "fim da gramatica" << endl;
	//operadores
	// cout << "inicio do operadores" << endl;
	// gram->imprimeGramatica();
	op = new Operators();
	// cout << "fim do operadores" << endl << "inicio dos database" << endl;
	//dados
	//    data = new Database("read/base5.txt", "read/grupo5.txt");
	Database *banco_dados = new Database(dados, grupo);
	Database* d_banco_dados =banco_dados->copyDevice();
	teste<<<1, 1>>>(d_banco_dados);
	//banco_dados->print();
	//cout << "fim do database" << endl << "inicio do search" << endl;
	//busca
	Search* s = new Search(banco_dados);
	delete s;

	fclose(stdout);

	return 0;
}

