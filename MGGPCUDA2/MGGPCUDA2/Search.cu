#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <iomanip>
#include "Search.h"
//#include "Population.h"
#include "Subject.h"
#include "Database.h"
#include "Gramatica.h"
#include "Configures.h"
#include "Parser.h"
#include "Device_Subject.h"


using namespace std;


class D_No {
public:
	double valor;
	D_No* next;
	__device__ D_No() {
		valor = 0.0;
		next = NULL;
	}
};

class Pilha {
public:
	D_No* atual;

__device__	Pilha() {
	atual = NULL;
	}

 __device__ ~Pilha() {
	if (atual == NULL) {
		return;
	}
	
	while (atual->next != NULL) {
		puxar();
	}
	puxar();
}
__device__	void push(double v) {
		if (atual == NULL) {
			//hipMalloc(&atual, sizeof(D_No));
			atual = new D_No();
			atual->valor = v;
		}
		else {
			D_No* aux;
			//hipMalloc(&aux, sizeof(D_No));
			aux = new D_No();
			aux->valor = v;
			aux->next = atual;
			atual = aux;
		}
}
__device__	void puxar() {
	if (atual != NULL) {
		D_No* aux = atual;
		//double res = atual->valor;
		atual = atual->next;
		//hipFree(aux);
		delete aux;
		//return res;
	}
	}
__device__ double top() {
	if (atual == NULL) {
		return 0.0;
	}
	return atual->valor;
}

};

__device__ double d_opera(double a, int x) {
	if (x == 0) //log
				//        return a >= 0 ? log(a) : INFINITY;
		return log(a);
	else if (x == 1) //exp
		return exp(a);
	else if (x == 2)//sqrt
					//        return a > 0 ? sqrt(a) : INFINITY;
		return sqrt(a);
}

__device__ double d_opera(double a, double b, int x) {
	if (x == 0)
		return a + b;
	else if (x == 1)
		return a - b;
	else if (x == 2)
		return a * b;
	else if (x == 3) {
		//        return b == 0 ? a/b : INFINITY;
		return a / b;
	}
	else if (x == 4)
		return pow(a, b);
	//        return a > 0 ? pow(a, b) : INFINITY;
	else if (x == 5)
		return a + b;
}

__device__ double d_operaLogic(double a, double b, int x) {
	if (x == 0)
		return a && b;
	else if (x == 1)
		return a || b;
	else if (x == 2)
		return (bool)a ^ (bool)b;
}
//7
__device__ double d_operaLogic(double a, int x) {
	if (x == 0)
		return !(bool)a;
	else if (x == 1)
		return 1;
	else if (x == 2)
		return 0;
}
//8
__device__ double d_operaComp(double a, double b, int x) {
	if (x == 0)
		return a < b;
	else if (x == 1)
		return a <= b;
	else if (x == 2)
		return a == b;
	else if (x == 3)
		return a >= b;
	else if (x == 4)
		return a > b;
	else if (x == 5)
		return a != b;
}
//9
__device__ double d_operaIfElse(double a, double b, int c) {
	if (a == 0) {
		return b;
	}
	else {
		return c;
	}
}


__device__ double avalia(double* expr, int expCounter) {
	
	
	Pilha q;
	int aux;
	//int countAlpha = 0;
	for (int i = 0; i < expCounter; i += 2) {
		double result = 0.0;
		switch ((int)expr[i]) {
		case 0:
		{
			q.push(expr[i + 1]);
		}
		break;
		case 1:// + - * / pow !
		{
			double b = q.top();
			q.puxar();
			double d = q.top();
			q.puxar();
			result = d_opera(d, b, expr[i + 1]);
			q.push(result);
		}
		break;
		case 2:// log exp sqrt
		{
			double n = q.top();
			q.puxar();
			result = d_opera(n, expr[i + 1]);
			q.push(result);
		}
		break;
		case 6:// && || xor
		{
			double a = q.top();
			q.puxar();
			double b = q.top();
			q.puxar();
			result = d_operaLogic(b, a, expr[i + 1]);
			q.push(result);
		}
		break;
		case 7:
		{
			double a = q.top();
			q.puxar();
			result = d_operaLogic(a, expr[i + 1]);
			q.push(result);
		}
		break;
		case 8:
		{
			double a = q.top();
			q.puxar();
			double b = q.top();
			q.puxar();
			result = d_operaComp(b, a, expr[i + 1]);
			q.push(result);
		}
		break;
		case 9:
		{
			double a = q.top();
			q.puxar();
			double b = q.top();
			q.puxar();
			double c = q.top();
			q.puxar();
			result = d_operaIfElse(a, b, c);
			q.push(result);
		}
		}
		if (isnan(result) || isinf(result)) {
			return INFINITY;
		}
	}

	double resultado = q.top();
	// resultado = 0.0;
	return resultado;
}

__device__ double treeResult(double* var,double* exp,int expCounter) {
	double result = 0;
	double* a = new double[expCounter];
	for (int i = 0; i < expCounter; i += 2) {
		a[i] = exp[i];
		a[i + 1] = exp[i + 1];
		if (a[i] == 3.0) {
			a[i] = 0.0;
			a[i + 1] = var[(int)a[i + 1]];
		}
		if (a[i] == 5.0) {
			a[i] = 0.0;
		}
	}

	result = avalia(a, expCounter);
	delete[] a;
	return result;
	//return var[0];

}



__global__ void kernelObj(Database* d_dados, Device_Subject** d_pop) {
	if (blockIdx.x < gridDim.x) {
		Device_Subject* d_ind = d_pop[blockIdx.x];
		d_ind->vp = d_ind->fp = d_ind->fn = d_ind->vn = 0;
		for (int i = 0; i < d_dados->trainCount; i++) {
			int id = d_dados->training[i];
			double yReal = d_dados->results[id];
			double yPredict = treeResult(d_dados->values[id], d_ind->d_tree_exp, d_ind->d_tree_countExp);
			if (yPredict != yReal) {
				if (yReal == 0.0) {
					d_ind->fp++;
				}
				else {
					d_ind->fn++;
				}

			}
			else {
				if (yReal == 0.0) {
					d_ind->vn++;

				}
				else {
					d_ind->vp++;
				}
			}
			
		}
	d_ind->erro = ((double)(d_ind->fn + d_ind->fp) / d_dados->trainCount) * 100;

	}

}


void Search::GPUcalcFitnessLS(int ini,int fim) {
	Device_Subject** d_pop;
	Device_Subject** aux;
	size_t tam2;
	int tamTreino = this->banco_dados->trainCount;
	int tamPop = h_conf->popSize;
	int tam = fim - ini;
	aux = new Device_Subject*[tam];
	tam2 = sizeof(Device_Subject*)*tam;
	hipSetDevice(0);
	//carregando na GPU
	for(int i = 0; i < tam; i++) {
		Device_Subject* sub = new Device_Subject();
		sub->iniDeviceTree(pop[i + ini]);
		
		hipMalloc(&aux[i], sizeof(Device_Subject));
		hipMemcpy(aux[i],sub,sizeof(Device_Subject),hipMemcpyHostToDevice);
		
	}

	hipMalloc(&d_pop, tam2);
	hipMemcpy(d_pop, aux, tam2, hipMemcpyHostToDevice);
	
	
	
	//executando
	
	//dim3 block(tamPop, tamTreino);
	dim3 block(tamPop);
	
	kernelObj<<<block, 1>>>(this->d_banco_dados, d_pop);
	Device_Subject** tst = new Device_Subject*[tam];
	hipMemcpy(tst, d_pop, tam2, hipMemcpyDeviceToHost);
	hipError_t erro = hipGetLastError();
	
	for (int i = 0; i < tam; i++) {
		Device_Subject novo;
		Subject* atual = pop[i + ini];
		hipMemcpy(&novo, aux[i], sizeof(Device_Subject), hipMemcpyDeviceToHost);
		atual->fitnessLS = novo.erro;
		atual->treino_vp = novo.vp;
		atual->treino_fp = novo.fp;
		atual->treino_fn = novo.fn;
		atual->treino_vn = novo.vn;
		novo.destDeviceTree();
		hipFree(aux[i]);
		atual->complexity();
	}
//descaregando da GPU
	
	hipFree(d_pop);
	
	delete[] aux;
}

bool mySort(Subject* a, Subject* b) {
	//    return (a->fitness < b->fitness);
	return (a->fitnessLS < b->fitnessLS);
};

bool sortTest(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessTestLS < b->fitnessTestLS);
};

bool sortRank(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->ranking < b->ranking);
};

bool Search::sortHigh(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->tree->high < b->tree->high);
};

bool Search::sortComplexity(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->complex < b->complex);
};

bool Search::sortSize(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->tree->terminals < b->tree->terminals);
};

bool Search::sortFit(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessLS < b->fitnessLS);
};

bool Search::sortFitTest(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessTestLS < b->fitnessTestLS);
};

bool Search::sortFitValid(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessValidLS < b->fitnessValidLS);
};

bool sortCrow(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->crowdingDistance > b->crowdingDistance);
};

Search::Search(Database* banco_dados, Database* d_banco) {
	testError = INFINITY;
	testTr = INFINITY;
	errors = 0;
	this->banco_dados = banco_dados;
	this->d_banco_dados = d_banco;
	
	aux = new int[h_conf->popSize];
	for (int i = 0; i < h_conf->popSize; i++)
		aux[i] = 0;

	h_conf->mono == 0 ? doMonoSearch() : doMultiSearch();
};

void Search::doMonoSearch() {
	pop = new Subject*[h_conf->popSize * 2];
	convergence = new double[h_conf->iterations];

	for (int i = 0; i < h_conf->popSize; i++) { //Cria popula��o
		pop[i] = new Subject(gram->criaArvExp());
	}

	for (int i = 0; i < h_conf->popSize; i++) //Calcula primeira vez
	{
		calcFitnessLS(pop[i]);
	}

	int size = (h_conf->popSize * h_conf->elitism);

	for (int it = 0; it < h_conf->iterations; it++) {
		Operate();
		sort(pop, pop + h_conf->popSize, sortFit);
		sort(pop + h_conf->popSize, pop + h_conf->popSize * 2, sortFit);

		sort(pop, pop + h_conf->popSize + size, sortFit);
		for (int i = h_conf->popSize; i < h_conf->popSize * 2; i++)
			delete pop[i];
		convergence[it] = pop[0]->fitnessLS;

		//        cout << it << endl;
		//        if(it > (int)(conf->iterations * 0.00005))
		//            cout << convergence[it]/convergence[(int)(it - conf->iterations * 0.00005)] << endl;
		//        if(it > (int)(conf->iterations * 0.00005) && convergence[it] > (convergence[(int)(it - conf->iterations * 0.00005)] - convergence[(int)(it - conf->iterations * 0.00005)] * 0.00000000001)){
		//            cout << "Stopped Converging on " << it << "!" << endl;
		//            break;
		//        }
	}

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessTestLS(pop[i]);
	}

	sort(pop, pop + h_conf->popSize, sortFitTest);

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessValidLS(pop[i]);
	}

	sort(pop, pop + h_conf->popSize, sortFitValid);

	for (int i = 0; i < h_conf->popSize; i++) {
		cout << pop[i]->ranking << ", "
			<< pop[i]->tree->high << ", "
			<< pop[i]->fitnessLS << ", "
			<< pop[i]->fitnessTestLS << ", "
			<< pop[i]->fitnessValidLS << ", "
			<< pop[i]->tree->infixPrint() << ", ";
		pop[i]->tree->printAlphas();
		cout << endl;
	}

	//    double diff = abs(convergence[conf->iterations - 1]) + 1;
	//    double max = convergence[conf->iterations - 1] < 0 ? convergence[0] + convergence[conf->iterations - 1] : convergence[0];
	//    cout << diff << " " << max << endl;
	//    cin.get();
	//
	//    for(int i = 0; i < conf->iterations; i += conf->iterations * .1){
	//        cout << convergence[i] << endl;
	////        for(int j = 0; j < convergence[i] + diff; j += max * .1){
	////            cout << "-";
	////        }
	////        cout << "." << endl;
	//    }
};

void Search::doMultiSearch() {
	pop = new Subject*[h_conf->popSize * 2];
	convergence = new double[h_conf->iterations];

	for (int i = 0; i < h_conf->popSize; i++) { //Cria popula��o
		pop[i] = new Subject(gram->criaArvExp());
	}

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessLS(pop[i]);
	}

	//    for(int i = 0; i < 100000000; i++)
	//        calcRank(conf->popSize);

	//    double old = 0;
	for (int it = 0; it < h_conf->iterations; it++) {
		Operate();
		int size = h_conf->popSize * 2;
		calcRank(size);
		// apagar piores
		for (int i = h_conf->popSize; i < size; i++)
			delete pop[i];

		// teste
		if (it % 10 == 0) {
			cout << it << "\t";
			for (int i = 0; i < h_conf->popSize; i++) {
				if (pop[i]->ranking == 0) {
					cout << pop[i]->tree->high << ","
						<< pop[i]->tree->terminals << ","
						<< pop[i]->fitnessLS;
					cout << ";";
				}

			}
			cout << endl;
		}
	}
	// fim teste
	cout << "RESULTADO" << endl;
	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessTestLS(pop[i]);
	}

	calcRankTest(h_conf->popSize);

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessValidLS(pop[i]);
	}

	calcRankValid(h_conf->popSize);

	for (int i = 0; i < h_conf->popSize; i++) {
		cout << pop[i]->ranking << ", "
			<< pop[i]->tree->high << ", "
			<< pop[i]->tree->terminals << ", "
			<< pop[i]->fitnessLS << ", "
			<< pop[i]->fitnessTestLS << ", "
			<< pop[i]->fitnessValidLS << ", "
			<< pop[i]->tree->infixPrint() << ", " << "pop[i]->tree->print()"
			<< ",( " << pop[i]->treino_vp << ";" << pop[i]->treino_vn << ";" << pop[i]->treino_fp << ";" << pop[i]->treino_fn << ")"
			<< ",( " << pop[i]->teste_vp << ";" << pop[i]->teste_vn << ";" << pop[i]->teste_fp << ";" << pop[i]->teste_fn << ")"
			<< ",( " << pop[i]->valida_vp << ";" << pop[i]->valida_vn << ";" << pop[i]->valida_fp << ";" << pop[i]->valida_fn << ")";
		//             pop[i]->tree->printAlphas();
		cout << endl;
	}
	// inserir modifica��o
	cout << "VETORES" << endl;
	cout << "TREINO" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->trainCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->training[l]], NULL, 0) << ";";
		}

		cout << endl;
	}
	cout << "TESTE" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->testCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->test[l]], NULL, 0) << ";";
		}

		cout << endl;
	}
	cout << "VALIDATION" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->validCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->validation[l]], NULL, 0) << ";";
		}
		cout << endl;
	}

	// Encontrado
	cout << "ENCONTRADO" << endl;
	for (int i = 0; i<banco_dados->trainCount; i++) {
		cout << banco_dados->results[banco_dados->training[i]] << ";";
	}
	cout << endl;
	for (int i = 0; i<banco_dados->testCount; i++) {
		cout << banco_dados->results[banco_dados->test[i]] << ";";
	}
	cout << endl;
	for (int i = 0; i<banco_dados->validCount; i++) {
		cout << banco_dados->results[banco_dados->validation[i]] << ";";
	}
};

int Search::tournamentMono(int a, int b) {
	if (pop[a]->fitnessLS < pop[b]->fitnessLS)
		return a;
	else
		return b;
}

int Search::tournamentMulti(int a, int b) {
	if (pop[a]->ranking < pop[b]->ranking)
		return a;
	else if (pop[a]->ranking > pop[b]->ranking)
		return b;
	else {
		if (pop[a]->crowdingDistance > pop[b]->crowdingDistance)
			return a;
		else
			return b;
	}
}

void Search::Operate() {
	//instanciar
	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i += 2) {
		pop[i] = (new Subject());
		pop[i + 1] = (new Subject());
	}
	//muta��o e cross
	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i += 2) {
		int a, b;
		int s1 = rand() % h_conf->popSize;
		int s2 = rand() % h_conf->popSize;
		a = h_conf->mono == 0 ? tournamentMono(s1, s2) : tournamentMulti(s1, s2);

		s1 = rand() % h_conf->popSize;
		s2 = rand() % h_conf->popSize;
		b = h_conf->mono == 0 ? tournamentMono(s1, s2) : tournamentMulti(s1, s2);

		aux[a]++;
		aux[b]++;

		op->Cross(pop[a]->tree, pop[b]->tree, pop[i]->tree, pop[i + 1]->tree);

		op->Mutate(pop[i]->tree);
		op->Mutate(pop[i + 1]->tree);
	}

	//calcFitnessLS 
	//paralelizar

	GPUcalcFitnessLS(h_conf->popSize, h_conf->popSize * 2);
	

	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i++) {
		calcFitnessLS(pop[i]);
	}
	
	
};

bool Search::dominate(Subject* a, Subject* b) {
	if (a->fitnessLS <= b->fitnessLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessLS < b->fitnessLS) {
			return true;
		}
	}
	return false;
};

bool Search::dominateTest(Subject* a, Subject* b) {
	if (a->fitnessTestLS <= b->fitnessTestLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessTestLS < b->fitnessTestLS) {
			return true;
		}
	}
	return false;
};

bool Search::dominateValid(Subject* a, Subject* b) {
	if (a->fitnessValidLS <= b->fitnessValidLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessValidLS < b->fitnessValidLS) {
			return true;
		}
	}
	return false;
};

void Search::calcRank(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						// if (pop[i]->tree->infixPrint().compare(pop[j]->tree->infixPrint()) == 0) {
						if (rand() % 100 < 50 && pop[i]->fitnessLS == pop[j]->fitnessLS && pop[i]->complex == pop[j]->complex) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominate(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}



					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFit);
		crowdingDistanceFitness(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		if (f > h_conf->popSize)
			sort(pop + i, pop + f, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::calcRankTest(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						if (pop[i]->tree->infixPrint() == pop[j]->tree->infixPrint()) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominateTest(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}
					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (f < h_conf->popSize && pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFitTest);
		crowdingDistanceFitnessTest(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		//desordena o rank se o tamanho dele for >= conf->popSize
		if (f > h_conf->popSize)
			sort(pop + i, pop + f - 1, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::calcRankValid(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						if (pop[i]->tree->infixPrint() == pop[j]->tree->infixPrint()) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominateValid(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}
					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (f < h_conf->popSize && pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFitValid);
		crowdingDistanceFitnessValid(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		if (f > h_conf->popSize)
			sort(pop + i, pop + f - 1, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::crowdingDistanceFitness(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessLS - pop[j - 1]->fitnessLS) / (pop[f]->fitnessLS - pop[i]->fitnessLS);
		}
	}
}

void Search::crowdingDistanceFitnessTest(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessTestLS - pop[j - 1]->fitnessTestLS) / (pop[f]->fitnessTestLS - pop[i]->fitnessTestLS);
		}
	}
}

void Search::crowdingDistanceFitnessValid(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = INFINITY;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessValidLS - pop[j - 1]->fitnessValidLS) / (pop[f]->fitnessValidLS - pop[i]->fitnessValidLS);
		}
	}
}

void Search::crowdingDistanceHigh(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->tree->high - pop[j - 1]->tree->high);
			double den = (pop[f]->tree->high - pop[i]->tree->high);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::crowdingDistanceComplexity(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->complex - pop[j - 1]->complex);
			double den = (pop[f]->complex - pop[i]->complex);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::crowdingDistanceSize(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->tree->terminals - pop[j - 1]->tree->terminals);
			double den = (pop[f]->tree->terminals - pop[i]->tree->terminals);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::calcFitnessLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->trainCount;
	// int columns = s->tree->subCounter;
	int dimension = 0;
	//   double** subMat;
	double* alpha = NULL;

	s->treino_vp = s->treino_fp = s->treino_vn = s->treino_fn = 0;

	//classification

	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->training[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->training[i]]) {
			if (banco_dados->results[banco_dados->training[i]] == 0) {
				//fitness += conf->peso0;
				s->treino_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->treino_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->training[i]] == 0) {
				s->treino_vn++;
			}
			else {
				s->treino_vp++;
			}
		}
	}

	s->fitnessLS = ((double)(s->treino_fn + s->treino_fp) / lines) * 100;
	s->complex = s->complexity();

	//    //regression
	//    if(conf->leastSquare == 1){
	//        dimension = columns;
	//        subMat = new double*[lines];
	//        for(int i = 0; i < lines; i++)
	//            subMat[i] = new double[columns];
	//
	//        //Avalia cada subExp
	//        for(int i = 0; i < columns; i++)
	//        {
	//            double* exp = &s->tree->sub.at(i).at(0);
	//            int sizeExp = s->tree->sub.at(i).size();
	//            for(int j = 0; j < lines; j++)
	//            {
	//                double* a = new double[sizeExp];
	//                double* var = data->values[data->training[j]];
	//                for(int k = 0; k < sizeExp; k += 2)
	//                {
	//                    a[k] = exp[k];
	//                    a[k + 1] = exp[k + 1];
	//                    if(a[k] == 3.0)
	//                    {
	//                        a[k] = 0.0;
	//                        a[k + 1] = var[(int)a[k + 1]];
	//                    }
	//                    if(a[k] == 5.0)
	//                    {
	//                        a[k] = 0.0;
	//                    }
	//                }
	//
	//                subMat[j][i] = Avalia(a, sizeExp);
	//                delete [] a;
	//            }
	//        }
	//
	//
	//        QRDecomposition* qrDec = new QRDecomposition(subMat, lines, columns);
	//        double* b = new double[lines];
	//        for(int i = 0; i < lines; i++)
	//        {
	//            b[i] = data->results[data->training[i]];
	//        }
	////        se tem algo l�, apagar
	//        alpha = s->tree->alpha;
	//        if(alpha != NULL)
	//            delete [] alpha;
	////        novos coeficientes
	//        alpha = s->tree->alpha = qrDec->solveLeastSquares(b, lines);
	//
	//        if(alpha == NULL){
	//            dimension = 0;
	//        }
	//
	//        for(int i = 0; i < lines; i++)
	//            delete [] subMat[i];
	//        delete [] subMat;
	//        delete [] b;
	//        delete qrDec;
	//    }
	//
	//    for(int i = 0; i < lines ; i++){
	//        double res = (s->tree->treeResult(data->values[data->training[i]], alpha, dimension));
	//
	////        for(int j = 0; j < data->countVar; j++)
	////            cout << data->values[data->training[i]][j] << " ";
	////        if(s->tree->infixPrint() == "(a + ((c * b) + (d / e)))"){
	////            cout << data->results[data->training[i]] << " " << res << " " << data->results[data->training[i]] - res << " " << pow(data->results[data->training[i]] - res, 2) << endl;
	////            cin.get();
	////        }
	//
	//        res = pow(data->results[data->training[i]] - res, 2);
	//
	//        if(res == INFINITY){
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN){
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessLS = fitness/lines;
};

void Search::calcFitnessTestLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->testCount;
	int columns = s->tree->subCounter;
	double* alpha = s->tree->alpha;
	int dimension = columns;
	if (h_conf->leastSquare == 0 || alpha == NULL) {
		dimension = 0;
	}

	s->teste_vp = s->teste_fp = s->teste_vn = s->teste_fn = 0;

	//classification
	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->test[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->test[i]]) {
			if (banco_dados->results[banco_dados->test[i]] == 0) {
				//fitness += conf->peso0;
				s->teste_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->teste_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->test[i]] == 0) {
				s->teste_vn++;
			}
			else {
				s->teste_vp++;
			}
		}
	}

	s->fitnessTestLS = ((double)(s->teste_fn + s->teste_fp) / lines) * 100;

	//    //regression
	//    for(int i = 0; i < lines ; i++)
	//    {
	//        double res = (s->tree->treeResult(data->values[data->test[i]], alpha, dimension));
	//        res = pow(data->results[data->test[i]] - res, 2);
	//        if(res == INFINITY)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessTestLS = fitness/lines;
};

void Search::calcFitnessValidLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->validCount;
	int columns = s->tree->subCounter;
	double* alpha = s->tree->alpha;
	int dimension = columns;
	if (h_conf->leastSquare == 0 || alpha == NULL) {
		dimension = 0;
	}

	s->valida_vp = s->valida_fp = s->valida_vn = s->valida_fn = 0;

	//classification
	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->validation[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->validation[i]]) {
			if (banco_dados->results[banco_dados->validation[i]] == 0) {
				//fitness += conf->peso0;
				s->valida_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->valida_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->validation[i]] == 0) {
				s->valida_vn++;
			}
			else {
				s->valida_vp++;
			}
		}
	}

	s->fitnessValidLS = ((double)(s->valida_fn + s->valida_fp) / lines) * 100;
	//      //regression
	//    for(int i = 0; i < lines ; i++)
	//    {
	//        double res = (s->tree->treeResult(data->values[data->validation[i]], alpha, dimension));
	//        res = pow(data->results[data->validation[i]] - res, 2);
	//        if(res == INFINITY)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessValidLS = fitness/lines;
};

void Search::Replace() {
	for (int i = h_conf->popSize * 2 - 1; i >= h_conf->popSize; i--)
		delete pop[i];
};

Search::~Search() {
	delete[] pop;
};
