#include "hip/hip_runtime.h"
#include <algorithm>
#include <math.h>
#include <iomanip>
#include "Search.h"
//#include "Population.h"
#include "Subject.h"
#include "Database.h"
#include "Gramatica.h"
#include "Configures.h"
#include "Parser.h"

using namespace std;

__global__ void teste(Database* d_dados, Configures* d_conf,Subject** d_pop) {
	d_dados->countVar = 20;
}

void Search::GPUcalcFitnessLS(int ini,int fim) {
	Subject** d_pop;
	Subject** aux;
	int tam = fim - ini;
	aux = new Subject*[tam];
	//carregando na GPU
	for (int i = 0; i < tam; i++) {
		//pop[i + ini]->iniDeviceTree();
		hipMalloc(&aux[i], sizeof(Subject));
		hipMemcpy(aux[i],pop[i+ini],sizeof(Subject),hipMemcpyHostToDevice);
	}

	hipMalloc(&d_pop, sizeof(Subject*)*tam);
	hipMemcpy(d_pop, aux, sizeof(Subject*)*tam, hipMemcpyHostToDevice);

	
	//executando
	teste<<<1, 1>>>(this->d_banco_dados, this->d_conf,d_pop);
	//descaregando da GPU
	for (int i = 0; i < tam; i++) {
		//pop[i + ini]->destDeviceTree();
		hipFree(&d_pop[i]);
	}
	hipFree(d_pop);
	delete aux;
}

bool mySort(Subject* a, Subject* b) {
	//    return (a->fitness < b->fitness);
	return (a->fitnessLS < b->fitnessLS);
};

bool sortTest(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessTestLS < b->fitnessTestLS);
};

bool sortRank(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->ranking < b->ranking);
};

bool Search::sortHigh(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->tree->high < b->tree->high);
};

bool Search::sortComplexity(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->complex < b->complex);
};

bool Search::sortSize(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->tree->terminals < b->tree->terminals);
};

bool Search::sortFit(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessLS < b->fitnessLS);
};

bool Search::sortFitTest(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessTestLS < b->fitnessTestLS);
};

bool Search::sortFitValid(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->fitnessValidLS < b->fitnessValidLS);
};

bool sortCrow(Subject* a, Subject* b) {
	//    return (a->fitnessTest < b->fitnessTest);
	return (a->crowdingDistance > b->crowdingDistance);
};

Search::Search(Database* banco_dados, Database* d_banco) {
	testError = INFINITY;
	testTr = INFINITY;
	errors = 0;
	this->banco_dados = banco_dados;
	this->d_banco_dados = d_banco;
	hipMalloc(&this->d_conf, sizeof(Configures));
	hipMemcpy(this->d_conf, h_conf, sizeof(Configures), hipMemcpyHostToDevice);
	aux = new int[h_conf->popSize];
	for (int i = 0; i < h_conf->popSize; i++)
		aux[i] = 0;

	h_conf->mono == 0 ? doMonoSearch() : doMultiSearch();
};

void Search::doMonoSearch() {
	pop = new Subject*[h_conf->popSize * 2];
	convergence = new double[h_conf->iterations];

	for (int i = 0; i < h_conf->popSize; i++) { //Cria popula��o
		pop[i] = new Subject(gram->criaArvExp());
	}

	for (int i = 0; i < h_conf->popSize; i++) //Calcula primeira vez
	{
		calcFitnessLS(pop[i]);
	}

	int size = (h_conf->popSize * h_conf->elitism);

	for (int it = 0; it < h_conf->iterations; it++) {
		Operate();
		sort(pop, pop + h_conf->popSize, sortFit);
		sort(pop + h_conf->popSize, pop + h_conf->popSize * 2, sortFit);

		sort(pop, pop + h_conf->popSize + size, sortFit);
		for (int i = h_conf->popSize; i < h_conf->popSize * 2; i++)
			delete pop[i];
		convergence[it] = pop[0]->fitnessLS;

		//        cout << it << endl;
		//        if(it > (int)(conf->iterations * 0.00005))
		//            cout << convergence[it]/convergence[(int)(it - conf->iterations * 0.00005)] << endl;
		//        if(it > (int)(conf->iterations * 0.00005) && convergence[it] > (convergence[(int)(it - conf->iterations * 0.00005)] - convergence[(int)(it - conf->iterations * 0.00005)] * 0.00000000001)){
		//            cout << "Stopped Converging on " << it << "!" << endl;
		//            break;
		//        }
	}

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessTestLS(pop[i]);
	}

	sort(pop, pop + h_conf->popSize, sortFitTest);

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessValidLS(pop[i]);
	}

	sort(pop, pop + h_conf->popSize, sortFitValid);

	for (int i = 0; i < h_conf->popSize; i++) {
		cout << pop[i]->ranking << ", "
			<< pop[i]->tree->high << ", "
			<< pop[i]->fitnessLS << ", "
			<< pop[i]->fitnessTestLS << ", "
			<< pop[i]->fitnessValidLS << ", "
			<< pop[i]->tree->infixPrint() << ", ";
		pop[i]->tree->printAlphas();
		cout << endl;
	}

	//    double diff = abs(convergence[conf->iterations - 1]) + 1;
	//    double max = convergence[conf->iterations - 1] < 0 ? convergence[0] + convergence[conf->iterations - 1] : convergence[0];
	//    cout << diff << " " << max << endl;
	//    cin.get();
	//
	//    for(int i = 0; i < conf->iterations; i += conf->iterations * .1){
	//        cout << convergence[i] << endl;
	////        for(int j = 0; j < convergence[i] + diff; j += max * .1){
	////            cout << "-";
	////        }
	////        cout << "." << endl;
	//    }
};

void Search::doMultiSearch() {
	pop = new Subject*[h_conf->popSize * 2];
	convergence = new double[h_conf->iterations];

	for (int i = 0; i < h_conf->popSize; i++) { //Cria popula��o
		pop[i] = new Subject(gram->criaArvExp());
	}

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessLS(pop[i]);
	}

	//    for(int i = 0; i < 100000000; i++)
	//        calcRank(conf->popSize);

	//    double old = 0;
	for (int it = 0; it < h_conf->iterations; it++) {
		Operate();
		int size = h_conf->popSize * 2;
		calcRank(size);
		// apagar piores
		for (int i = h_conf->popSize; i < size; i++)
			delete pop[i];

		// teste
		if (it % 10 == 0) {
			cout << it << "\t";
			for (int i = 0; i < h_conf->popSize; i++) {
				if (pop[i]->ranking == 0) {
					cout << pop[i]->tree->high << ","
						<< pop[i]->tree->terminals << ","
						<< pop[i]->fitnessLS;
					cout << ";";
				}

			}
			cout << endl;
		}
	}
	// fim teste
	cout << "RESULTADO" << endl;
	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessTestLS(pop[i]);
	}

	calcRankTest(h_conf->popSize);

	for (int i = 0; i < h_conf->popSize; i++) {
		calcFitnessValidLS(pop[i]);
	}

	calcRankValid(h_conf->popSize);

	for (int i = 0; i < h_conf->popSize; i++) {
		cout << pop[i]->ranking << ", "
			<< pop[i]->tree->high << ", "
			<< pop[i]->tree->terminals << ", "
			<< pop[i]->fitnessLS << ", "
			<< pop[i]->fitnessTestLS << ", "
			<< pop[i]->fitnessValidLS << ", "
			<< pop[i]->tree->infixPrint() << ", " << "pop[i]->tree->print()"
			<< ",( " << pop[i]->treino_vp << ";" << pop[i]->treino_vn << ";" << pop[i]->treino_fp << ";" << pop[i]->treino_fn << ")"
			<< ",( " << pop[i]->teste_vp << ";" << pop[i]->teste_vn << ";" << pop[i]->teste_fp << ";" << pop[i]->teste_fn << ")"
			<< ",( " << pop[i]->valida_vp << ";" << pop[i]->valida_vn << ";" << pop[i]->valida_fp << ";" << pop[i]->valida_fn << ")";
		//             pop[i]->tree->printAlphas();
		cout << endl;
	}
	// inserir modifica��o
	cout << "VETORES" << endl;
	cout << "TREINO" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->trainCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->training[l]], NULL, 0) << ";";
		}

		cout << endl;
	}
	cout << "TESTE" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->testCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->test[l]], NULL, 0) << ";";
		}

		cout << endl;
	}
	cout << "VALIDATION" << endl;
	for (int i = 0; i<h_conf->popSize; i++) {
		Subject* s = pop[i];
		for (int l = 0; l<banco_dados->validCount; l++) {

			cout << s->tree->treeResult(banco_dados->values[banco_dados->validation[l]], NULL, 0) << ";";
		}
		cout << endl;
	}

	// Encontrado
	cout << "ENCONTRADO" << endl;
	for (int i = 0; i<banco_dados->trainCount; i++) {
		cout << banco_dados->results[banco_dados->training[i]] << ";";
	}
	cout << endl;
	for (int i = 0; i<banco_dados->testCount; i++) {
		cout << banco_dados->results[banco_dados->test[i]] << ";";
	}
	cout << endl;
	for (int i = 0; i<banco_dados->validCount; i++) {
		cout << banco_dados->results[banco_dados->validation[i]] << ";";
	}
};

int Search::tournamentMono(int a, int b) {
	if (pop[a]->fitnessLS < pop[b]->fitnessLS)
		return a;
	else
		return b;
}

int Search::tournamentMulti(int a, int b) {
	if (pop[a]->ranking < pop[b]->ranking)
		return a;
	else if (pop[a]->ranking > pop[b]->ranking)
		return b;
	else {
		if (pop[a]->crowdingDistance > pop[b]->crowdingDistance)
			return a;
		else
			return b;
	}
}

void Search::Operate() {
	//instanciar
	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i += 2) {
		pop[i] = (new Subject());
		pop[i + 1] = (new Subject());
	}
	//muta��o e cross
	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i += 2) {
		int a, b;
		int s1 = rand() % h_conf->popSize;
		int s2 = rand() % h_conf->popSize;
		a = h_conf->mono == 0 ? tournamentMono(s1, s2) : tournamentMulti(s1, s2);

		s1 = rand() % h_conf->popSize;
		s2 = rand() % h_conf->popSize;
		b = h_conf->mono == 0 ? tournamentMono(s1, s2) : tournamentMulti(s1, s2);

		aux[a]++;
		aux[b]++;

		op->Cross(pop[a]->tree, pop[b]->tree, pop[i]->tree, pop[i + 1]->tree);

		op->Mutate(pop[i]->tree);
		op->Mutate(pop[i + 1]->tree);
	}

	//calcFitnessLS 
	//paralelizar

	//GPUcalcFitnessLS(h_conf->popSize, h_conf->popSize * 2);
	GPUcalcFitnessLS(0,h_conf->popSize);

	for (int i = h_conf->popSize; i < h_conf->popSize * 2; i++) {
		calcFitnessLS(pop[i]);
	}

	
};

bool Search::dominate(Subject* a, Subject* b) {
	if (a->fitnessLS <= b->fitnessLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessLS < b->fitnessLS) {
			return true;
		}
	}
	return false;
};

bool Search::dominateTest(Subject* a, Subject* b) {
	if (a->fitnessTestLS <= b->fitnessTestLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessTestLS < b->fitnessTestLS) {
			return true;
		}
	}
	return false;
};

bool Search::dominateValid(Subject* a, Subject* b) {
	if (a->fitnessValidLS <= b->fitnessValidLS && a->complex <= b->complex) {
		if (a->complex < b->complex) {
			return true;
		}
		else if (a->fitnessValidLS < b->fitnessValidLS) {
			return true;
		}
	}
	return false;
};

void Search::calcRank(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						// if (pop[i]->tree->infixPrint().compare(pop[j]->tree->infixPrint()) == 0) {
						if (rand() % 100 < 50 && pop[i]->fitnessLS == pop[j]->fitnessLS && pop[i]->complex == pop[j]->complex) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominate(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}



					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFit);
		crowdingDistanceFitness(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		if (f > h_conf->popSize)
			sort(pop + i, pop + f, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::calcRankTest(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						if (pop[i]->tree->infixPrint() == pop[j]->tree->infixPrint()) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominateTest(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}
					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (f < h_conf->popSize && pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFitTest);
		crowdingDistanceFitnessTest(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		//desordena o rank se o tamanho dele for >= conf->popSize
		if (f > h_conf->popSize)
			sort(pop + i, pop + f - 1, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::calcRankValid(int size) {
	/**
	Juntar tudo
	selecionar 1s n�o dominados rank = 0
	aumentar rank dos outros

	**/

	int ranking = 0;

	for (int i = 0; i < size; i++)
		pop[i]->ranking = 0;

	bool stop = false;
	while (!stop) {
		stop = true;
		for (int i = 0; i < size; i++) // ele
		{
			if (pop[i]->ranking == ranking) // se tiver na hora dele
			{
				for (int j = 0; j < size; j++) // pra cada um candidato
				{
					if (i != j && (pop[j]->ranking == ranking)) // se n�o for ele e se tiver no mesmo ranking
					{
						if (pop[i]->tree->infixPrint() == pop[j]->tree->infixPrint()) {
							pop[j]->ranking = 10000000;
							pop[j]->complex = 10000000;
							pop[j]->fitnessLS = 10000000;
						}
						else if (dominateValid(pop[i], pop[j])) // se for dominado por j ent�o ele aumenta o ranking
						{
							pop[j]->ranking = ranking + 1;
							stop = false;
						}
					}
				}
			}
		}
		ranking++;
	}
	sort(pop, pop + size, sortRank);

	int i, f, r;
	i = f = r = 0;
	while (f <= h_conf->popSize) {
		while (f < h_conf->popSize && pop[f]->ranking == r)
			f++;

		sort(pop + i, pop + f, sortFitValid);
		crowdingDistanceFitnessValid(i, f - 1);
		sort(pop + i, pop + f, sortComplexity);
		crowdingDistanceComplexity(i, f - 1);

		if (f > h_conf->popSize)
			sort(pop + i, pop + f - 1, sortCrow);

		i = f;
		f++;
		r++;
	}
};

void Search::crowdingDistanceFitness(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessLS - pop[j - 1]->fitnessLS) / (pop[f]->fitnessLS - pop[i]->fitnessLS);
		}
	}
}

void Search::crowdingDistanceFitnessTest(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessTestLS - pop[j - 1]->fitnessTestLS) / (pop[f]->fitnessTestLS - pop[i]->fitnessTestLS);
		}
	}
}

void Search::crowdingDistanceFitnessValid(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = INFINITY;
		for (int j = i + 1; j < f; j++) {
			pop[j]->crowdingDistance = (pop[j + 1]->fitnessValidLS - pop[j - 1]->fitnessValidLS) / (pop[f]->fitnessValidLS - pop[i]->fitnessValidLS);
		}
	}
}

void Search::crowdingDistanceHigh(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->tree->high - pop[j - 1]->tree->high);
			double den = (pop[f]->tree->high - pop[i]->tree->high);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::crowdingDistanceComplexity(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->complex - pop[j - 1]->complex);
			double den = (pop[f]->complex - pop[i]->complex);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::crowdingDistanceSize(int i, int f) {
	if (i == f) {
		pop[i]->crowdingDistance = MAX;
	}
	else {
		pop[i]->crowdingDistance = pop[f]->crowdingDistance = MAX;
		for (int j = i + 1; j < f; j++) {
			double num = (pop[j + 1]->tree->terminals - pop[j - 1]->tree->terminals);
			double den = (pop[f]->tree->terminals - pop[i]->tree->terminals);
			if (den == 0)
				pop[j]->crowdingDistance += 0;
			else
				pop[j]->crowdingDistance += num / den;
		}
	}
}

void Search::calcFitnessLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->trainCount;
	// int columns = s->tree->subCounter;
	int dimension = 0;
	//   double** subMat;
	double* alpha = NULL;

	s->treino_vp = s->treino_fp = s->treino_vn = s->treino_fn = 0;

	//classification

	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->training[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->training[i]]) {
			if (banco_dados->results[banco_dados->training[i]] == 0) {
				//fitness += conf->peso0;
				s->treino_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->treino_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->training[i]] == 0) {
				s->treino_vn++;
			}
			else {
				s->treino_vp++;
			}
		}
	}

	s->fitnessLS = ((double)(s->treino_fn + s->treino_fp) / lines) * 100;
	s->complex = s->complexity();

	//    //regression
	//    if(conf->leastSquare == 1){
	//        dimension = columns;
	//        subMat = new double*[lines];
	//        for(int i = 0; i < lines; i++)
	//            subMat[i] = new double[columns];
	//
	//        //Avalia cada subExp
	//        for(int i = 0; i < columns; i++)
	//        {
	//            double* exp = &s->tree->sub.at(i).at(0);
	//            int sizeExp = s->tree->sub.at(i).size();
	//            for(int j = 0; j < lines; j++)
	//            {
	//                double* a = new double[sizeExp];
	//                double* var = data->values[data->training[j]];
	//                for(int k = 0; k < sizeExp; k += 2)
	//                {
	//                    a[k] = exp[k];
	//                    a[k + 1] = exp[k + 1];
	//                    if(a[k] == 3.0)
	//                    {
	//                        a[k] = 0.0;
	//                        a[k + 1] = var[(int)a[k + 1]];
	//                    }
	//                    if(a[k] == 5.0)
	//                    {
	//                        a[k] = 0.0;
	//                    }
	//                }
	//
	//                subMat[j][i] = Avalia(a, sizeExp);
	//                delete [] a;
	//            }
	//        }
	//
	//
	//        QRDecomposition* qrDec = new QRDecomposition(subMat, lines, columns);
	//        double* b = new double[lines];
	//        for(int i = 0; i < lines; i++)
	//        {
	//            b[i] = data->results[data->training[i]];
	//        }
	////        se tem algo l�, apagar
	//        alpha = s->tree->alpha;
	//        if(alpha != NULL)
	//            delete [] alpha;
	////        novos coeficientes
	//        alpha = s->tree->alpha = qrDec->solveLeastSquares(b, lines);
	//
	//        if(alpha == NULL){
	//            dimension = 0;
	//        }
	//
	//        for(int i = 0; i < lines; i++)
	//            delete [] subMat[i];
	//        delete [] subMat;
	//        delete [] b;
	//        delete qrDec;
	//    }
	//
	//    for(int i = 0; i < lines ; i++){
	//        double res = (s->tree->treeResult(data->values[data->training[i]], alpha, dimension));
	//
	////        for(int j = 0; j < data->countVar; j++)
	////            cout << data->values[data->training[i]][j] << " ";
	////        if(s->tree->infixPrint() == "(a + ((c * b) + (d / e)))"){
	////            cout << data->results[data->training[i]] << " " << res << " " << data->results[data->training[i]] - res << " " << pow(data->results[data->training[i]] - res, 2) << endl;
	////            cin.get();
	////        }
	//
	//        res = pow(data->results[data->training[i]] - res, 2);
	//
	//        if(res == INFINITY){
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN){
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessLS = fitness/lines;
};

void Search::calcFitnessTestLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->testCount;
	int columns = s->tree->subCounter;
	double* alpha = s->tree->alpha;
	int dimension = columns;
	if (h_conf->leastSquare == 0 || alpha == NULL) {
		dimension = 0;
	}

	s->teste_vp = s->teste_fp = s->teste_vn = s->teste_fn = 0;

	//classification
	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->test[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->test[i]]) {
			if (banco_dados->results[banco_dados->test[i]] == 0) {
				//fitness += conf->peso0;
				s->teste_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->teste_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->test[i]] == 0) {
				s->teste_vn++;
			}
			else {
				s->teste_vp++;
			}
		}
	}

	s->fitnessTestLS = ((double)(s->teste_fn + s->teste_fp) / lines) * 100;

	//    //regression
	//    for(int i = 0; i < lines ; i++)
	//    {
	//        double res = (s->tree->treeResult(data->values[data->test[i]], alpha, dimension));
	//        res = pow(data->results[data->test[i]] - res, 2);
	//        if(res == INFINITY)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessTestLS = fitness/lines;
};

void Search::calcFitnessValidLS(Subject* s) {
	//double fitness = 0;
	int lines = banco_dados->validCount;
	int columns = s->tree->subCounter;
	double* alpha = s->tree->alpha;
	int dimension = columns;
	if (h_conf->leastSquare == 0 || alpha == NULL) {
		dimension = 0;
	}

	s->valida_vp = s->valida_fp = s->valida_vn = s->valida_fn = 0;

	//classification
	//double total = 0;

	for (int i = 0; i < lines; i++) {
		double res = (s->tree->treeResult(banco_dados->values[banco_dados->validation[i]], alpha, dimension));
		//        if (data->results[data->training[i]] == 0) {
		//            total += conf->peso0;
		//        } else {
		//            total += conf->peso1;
		//        }
		if (res != banco_dados->results[banco_dados->validation[i]]) {
			if (banco_dados->results[banco_dados->validation[i]] == 0) {
				//fitness += conf->peso0;
				s->valida_fp++;
			}
			else {
				//fitness += conf->peso1;
				s->valida_fn++;
			}
		}
		else {
			if (banco_dados->results[banco_dados->validation[i]] == 0) {
				s->valida_vn++;
			}
			else {
				s->valida_vp++;
			}
		}
	}

	s->fitnessValidLS = ((double)(s->valida_fn + s->valida_fp) / lines) * 100;
	//      //regression
	//    for(int i = 0; i < lines ; i++)
	//    {
	//        double res = (s->tree->treeResult(data->values[data->validation[i]], alpha, dimension));
	//        res = pow(data->results[data->validation[i]] - res, 2);
	//        if(res == INFINITY)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        if(res == NAN)
	//        {
	//            fitness = INFINITY;
	//            break;
	//        }
	//        fitness += res;
	//    }
	//    s->fitnessValidLS = fitness/lines;
};

void Search::Replace() {
	for (int i = h_conf->popSize * 2 - 1; i >= h_conf->popSize; i--)
		delete pop[i];
};

Search::~Search() {
	delete[] pop;
};
