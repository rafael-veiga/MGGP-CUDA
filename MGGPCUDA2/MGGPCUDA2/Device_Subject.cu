#include "hip/hip_runtime.h"
#include "Configures.h"
#include "Device_Subject.h"

Device_Subject::Device_Subject() {
	
};

void Device_Subject::iniDeviceTree( Subject* sub) {
	
	
	size_t tam = sizeof(double)*sub->tree->expCounter;
	this->d_tree_countExp = sub->tree->expCounter;
	hipMalloc(&this->d_tree_exp, tam);
	hipMemcpy(this->d_tree_exp, sub->tree->exp, tam, hipMemcpyHostToDevice);
	

}

void Device_Subject::destDeviceTree() {
	
	hipFree(this->d_tree_exp);
}