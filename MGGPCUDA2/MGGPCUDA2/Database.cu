#include "Database.h"

//Database* data;

char** Database::arrayStringAlocate(string* arraystr) {
	char** saida;
	char** aux = (char**) malloc(sizeof(char*)*arraystr->length());
	for (int i = 0; i < arraystr->length(); i++) {
		hipMalloc(&aux[i],sizeof(char)*30);
		hipMemcpy(aux[i], arraystr[i].c_str(), sizeof(char) * 30, hipMemcpyHostToDevice);
	}
	hipMalloc(&saida, sizeof(char*)*arraystr->length());
	hipMemcpy(saida, aux, sizeof(char*)*arraystr->length(), hipMemcpyHostToDevice);
	free(aux);
	return saida;
}

void Database::destroiArrayString(char** arrayString,int size) {
	for (int i = 0; i < size; i++) {
		hipFree(arrayString[i]);
	}
	hipFree(arrayString);
}

Database* Database::copyDevice() {
	Database* device;
	this->d_vars = arrayStringAlocate(this->vars);
	hipMalloc(&device, sizeof(Database));
	hipMemcpy(device, this, sizeof(Database), hipMemcpyHostToDevice);
	
	return device;
}

Database::Database(string base, string groups) {
	loadBase(base);
	loadGroups(groups);
};

void Database::loadBase(string base) {
	ifstream arq;
	string line;

	arq.open(base.c_str());

	arq >> countVar >> countResults;
	vars = new string[countVar];
	values = new double*[countResults];
	for (int i = 0; i < countResults; i++)
		values[i] = new double[countVar];
	results = new double[countResults];

	//inserido


	for (int i = 0; i < countVar; i++)
		arq >> vars[i];

	arq >> line;

	for (int i = 0; i < countResults; i++) {
		for (int j = 0; j < countVar; j++) {
			arq >> values[i][j];
		}
		arq >> results[i];
	}
	arq.close();
}

void Database::loadGroups(string groups) {
	//    ifstream arq;
	//    string line;
	//
	//    for(int i = 0; i < countVar; i++){
	//        int pos = gram->checkMapV(vars[i]);
	//        if(pos != -1){
	//            for(int j = 0; j < countResults; j++){
	//                swap(values[j][pos], values[j][i]);
	//            }
	//            swap(vars[i], vars[pos]);
	//        }
	//    }
	ifstream arq;
	string line;
	//int temp[30];

	int *temp = new int[countVar];
	for (int i = 0; i < countVar; i++)
		temp[i] = gram->checkMapV(vars[i]);


	for (int i = 0; i < countVar; i++) {
		int pos = temp[i];
		if (pos != -1) {
			for (int j = 0; j < countResults; j++) {
				swap(values[j][pos], values[j][i]);
			}
			swap(vars[i], vars[pos]);
			swap(temp[i], temp[pos]);
		}
	}

	arq.open(groups.c_str());
	arq >> trainCount >> testCount >> validCount;

	//conf->peso1 = 1.0 - conf->peso0;

	training = new int[trainCount];
	test = new int[testCount];
	validation = new int[validCount];

	arq >> line;
	for (int i = 0; i < trainCount; i++) {
		arq >> training[i];
	}
	arq >> line;
	for (int i = 0; i < testCount; i++) {
		arq >> test[i];
	}
	arq >> line;
	for (int i = 0; i < validCount; i++) {
		arq >> validation[i];
	}
	delete temp;
}

void Database::print() {
	cout << "vars: ";
	for (int i = 0; i < countVar; i++)
		cout << vars[i] << "  ";
	cout << endl;
	for (int i = 0; i < trainCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[training[i]][j] << "  ";
		cout << "   " << results[training[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < testCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[test[i]][j] << "  ";
		cout << "   " << results[test[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < validCount; i++) {
		cout << i << "   " << validation[i] << "  ";
		for (int j = 0; j < countVar; j++)
			cout << values[validation[i]][j] << "  ";
		cout << "   " << results[validation[i]] << endl;
	}
	cout << endl;
}

double* Database::getVars(int position) {
	return values[position];
}

