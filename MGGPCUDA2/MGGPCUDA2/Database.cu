#include "Database.h"

//Database* data;

//Database* Database::clone() {
//	Database* novo = new Database();
//	novo->countResults = this->countResults;
//	novo->countTestValues = this->countTestValues;
//	novo->countVar = this->countVar;
//	novo->testCount = this->testCount;
//	novo->trainCount = this->trainCount;
//	novo->validCount = this->validCount;
//}


Database* Database::copyDevice() {
	Database aux;
	Database* device;
	aux.countResults = this->countResults;
	aux.countTestValues = this->countTestValues;
	aux.countVar = this->countVar;
	aux.testCount = this->testCount;
	aux.trainCount = this->trainCount;
	aux.validCount = this->validCount;
	//alocar valores
	hipMalloc(&aux.values, sizeof(double*)*aux.countResults);
	
	double** inter = new double*[aux.countResults];
	for (int i = 0; i < aux.countResults; i++) {
		hipMalloc(&inter[i], sizeof(double)*aux.countVar);
		hipMemcpy(inter[i], this->values[i], sizeof(double)*aux.countVar, hipMemcpyHostToDevice);
	}
	
	hipMemcpy(aux.values, inter, sizeof(double**)*aux.countResults,hipMemcpyHostToDevice);
	delete inter;
	

	//alocar resultados
	hipMalloc(&aux.results, sizeof(double)*aux.countResults);
	hipMemcpy(aux.results, this->results, sizeof(double)*aux.countResults, hipMemcpyHostToDevice);

	//alocar treino
	hipMalloc(&aux.training, sizeof(int)*aux.trainCount);
	hipMemcpy(aux.training, this->training, sizeof(int)*aux.trainCount, hipMemcpyHostToDevice);

	//alocar teste
	hipMalloc(&aux.test, sizeof(int)*aux.testCount);
	hipMemcpy(aux.test, this->test, sizeof(int)*aux.testCount, hipMemcpyHostToDevice);

	//alocar valida
	hipMalloc(&aux.validation, sizeof(int)*aux.validCount);
	hipMemcpy(aux.validation, this->validation, sizeof(int)*aux.validCount, hipMemcpyHostToDevice);

	//carregando objeto
	hipMalloc(&device, sizeof(Database));
	hipMemcpy(device, &aux, sizeof(Database), hipMemcpyHostToDevice);

	return device;
}

Database::Database(string base, string groups) {
	loadBase(base);
	loadGroups(groups);
};

void Database::loadBase(string base) {
	ifstream arq;
	string line;

	arq.open(base.c_str());

	arq >> countVar >> countResults;
	vars = new string[countVar];
	values = new double*[countResults];
	for (int i = 0; i < countResults; i++)
		values[i] = new double[countVar];
	results = new double[countResults];

	//inserido


	for (int i = 0; i < countVar; i++)
		arq >> vars[i];

	arq >> line;

	for (int i = 0; i < countResults; i++) {
		for (int j = 0; j < countVar; j++) {
			arq >> values[i][j];
		}
		arq >> results[i];
	}
	arq.close();
}

void Database::loadGroups(string groups) {
	//    ifstream arq;
	//    string line;
	//
	//    for(int i = 0; i < countVar; i++){
	//        int pos = gram->checkMapV(vars[i]);
	//        if(pos != -1){
	//            for(int j = 0; j < countResults; j++){
	//                swap(values[j][pos], values[j][i]);
	//            }
	//            swap(vars[i], vars[pos]);
	//        }
	//    }
	ifstream arq;
	string line;
	//int temp[30];

	int *temp = new int[countVar];
	for (int i = 0; i < countVar; i++)
		temp[i] = gram->checkMapV(vars[i]);


	for (int i = 0; i < countVar; i++) {
		int pos = temp[i];
		if (pos != -1) {
			for (int j = 0; j < countResults; j++) {
				swap(values[j][pos], values[j][i]);
			}
			swap(vars[i], vars[pos]);
			swap(temp[i], temp[pos]);
		}
	}

	arq.open(groups.c_str());
	arq >> trainCount >> testCount >> validCount;

	//conf->peso1 = 1.0 - conf->peso0;

	training = new int[trainCount];
	test = new int[testCount];
	validation = new int[validCount];

	arq >> line;
	for (int i = 0; i < trainCount; i++) {
		arq >> training[i];
	}
	arq >> line;
	for (int i = 0; i < testCount; i++) {
		arq >> test[i];
	}
	arq >> line;
	for (int i = 0; i < validCount; i++) {
		arq >> validation[i];
	}
	delete temp;
}

void Database::print() {
	cout << "vars: ";
	for (int i = 0; i < countVar; i++)
		cout << vars[i] << "  ";
	cout << endl;
	for (int i = 0; i < trainCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[training[i]][j] << "  ";
		cout << "   " << results[training[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < testCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[test[i]][j] << "  ";
		cout << "   " << results[test[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < validCount; i++) {
		cout << i << "   " << validation[i] << "  ";
		for (int j = 0; j < countVar; j++)
			cout << values[validation[i]][j] << "  ";
		cout << "   " << results[validation[i]] << endl;
	}
	cout << endl;
}

double* Database::getVars(int position) {
	return values[position];
}

