#include "Database.h"

//Database* data;

Database* Database::copyDevice() {
	Database* device;
	hipMalloc(&device, sizeof(Database));
	hipMemcpy(device, this, sizeof(Database), hipMemcpyHostToDevice);
	hipMalloc(&device->vars, sizeof(string)*this->countVar);
	hipMemcpy(device->vars, this->vars, sizeof(string)*this->countVar, hipMemcpyHostToDevice);
	return device;
}

Database::Database(string base, string groups) {
	loadBase(base);
	loadGroups(groups);
};

void Database::loadBase(string base) {
	ifstream arq;
	string line;

	arq.open(base.c_str());

	arq >> countVar >> countResults;
	vars = new string[countVar];
	values = new double*[countResults];
	for (int i = 0; i < countResults; i++)
		values[i] = new double[countVar];
	results = new double[countResults];

	//inserido


	for (int i = 0; i < countVar; i++)
		arq >> vars[i];

	arq >> line;

	for (int i = 0; i < countResults; i++) {
		for (int j = 0; j < countVar; j++) {
			arq >> values[i][j];
		}
		arq >> results[i];
	}
	arq.close();
}

void Database::loadGroups(string groups) {
	//    ifstream arq;
	//    string line;
	//
	//    for(int i = 0; i < countVar; i++){
	//        int pos = gram->checkMapV(vars[i]);
	//        if(pos != -1){
	//            for(int j = 0; j < countResults; j++){
	//                swap(values[j][pos], values[j][i]);
	//            }
	//            swap(vars[i], vars[pos]);
	//        }
	//    }
	ifstream arq;
	string line;
	//int temp[30];

	int *temp = new int[countVar];
	for (int i = 0; i < countVar; i++)
		temp[i] = gram->checkMapV(vars[i]);


	for (int i = 0; i < countVar; i++) {
		int pos = temp[i];
		if (pos != -1) {
			for (int j = 0; j < countResults; j++) {
				swap(values[j][pos], values[j][i]);
			}
			swap(vars[i], vars[pos]);
			swap(temp[i], temp[pos]);
		}
	}

	arq.open(groups.c_str());
	arq >> trainCount >> testCount >> validCount;

	//conf->peso1 = 1.0 - conf->peso0;

	training = new int[trainCount];
	test = new int[testCount];
	validation = new int[validCount];

	arq >> line;
	for (int i = 0; i < trainCount; i++) {
		arq >> training[i];
	}
	arq >> line;
	for (int i = 0; i < testCount; i++) {
		arq >> test[i];
	}
	arq >> line;
	for (int i = 0; i < validCount; i++) {
		arq >> validation[i];
	}
	delete temp;
}

void Database::print() {
	cout << "vars: ";
	for (int i = 0; i < countVar; i++)
		cout << vars[i] << "  ";
	cout << endl;
	for (int i = 0; i < trainCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[training[i]][j] << "  ";
		cout << "   " << results[training[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < testCount; i++) {
		cout << i << "   ";
		for (int j = 0; j < countVar; j++)
			cout << values[test[i]][j] << "  ";
		cout << "   " << results[test[i]] << endl;
	}
	cout << endl;
	for (int i = 0; i < validCount; i++) {
		cout << i << "   " << validation[i] << "  ";
		for (int j = 0; j < countVar; j++)
			cout << values[validation[i]][j] << "  ";
		cout << "   " << results[validation[i]] << endl;
	}
	cout << endl;
}

double* Database::getVars(int position) {
	return values[position];
}

