#include <iostream>
#include <cmath>
#include "Configures.h"
#include "Subject.h"
#include "Tree.h"



Subject::Subject() {
	tree = new Tree();
	//    fitness = new double[objectives];
	//    fitnessTest = new double[objectives];
	fitnessLS = INFINITY;
	fitnessTestLS = INFINITY;
	printing = true;
};

//void Subject::iniDeviceTree() {
//	size_t tam = sizeof(Tree);
//	hipMalloc(&this->d_tree, tam);
//	hipMemcpy(this->d_tree, this->tree, tam, hipMemcpyHostToDevice);
//
//
//}
//
//void Subject::destDeviceTree() {
//	hipFree(this->d_tree);
//}

Subject::Subject(Tree* n) {
	tree = n;
	fitnessLS = INFINITY;
	fitnessTestLS = INFINITY;
	printing = true;
};

Subject::~Subject() {
	//    tree->print();
	delete tree;
};

double Subject::complexity() {
	if (h_conf->complexity == 0)
		return tree->high;
	else if (h_conf->complexity == 1)
		return tree->terminals;
}

void Subject::print() {
	//    cout << ".";
	tree->print();
};
