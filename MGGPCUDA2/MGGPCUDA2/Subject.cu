#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include "Configures.h"
#include "Subject.h"
#include "Tree.h"



Subject::Subject() {
	tree = new Tree();
	//    fitness = new double[objectives];
	//    fitnessTest = new double[objectives];
	fitnessLS = INFINITY;
	fitnessTestLS = INFINITY;
	printing = true;
};

void Subject::iniDeviceTree() {
	Device_Tree host;
	size_t tam = sizeof(double)*host.expCounter;
	host.expCounter = this->tree->expCounter;
	hipMalloc(&host.exp, tam);
	hipMemcpy(host.exp, this->tree->exp, tam, hipMemcpyHostToDevice);
	tam = sizeof(Device_Tree);
	hipMalloc(&this->d_tree, tam);
	hipMemcpy(this->d_tree, &host, tam, hipMemcpyHostToDevice);


}

void Subject::destDeviceTree() {
	hipFree(this->d_tree_exp);
	hipFree(this->d_tree);
}

Subject::Subject(Tree* n) {
	tree = n;
	fitnessLS = INFINITY;
	fitnessTestLS = INFINITY;
	printing = true;
};

Subject::~Subject() {
	//    tree->print();
	delete tree;
};

double Subject::complexity() {
	if (h_conf->complexity == 0)
		return tree->high;
	else if (h_conf->complexity == 1)
		return tree->terminals;
}

void Subject::print() {
	//    cout << ".";
	tree->print();
};
